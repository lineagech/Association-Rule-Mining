
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <pthread.h>
#include <assert.h>
#include <unistd.h>

#include <hip/hip_runtime_api.h>

#include <vector>
#include <unordered_map>
#include <iostream>
#include <fstream>
#include <numeric>
#include <functional>
#include <set>
#include <chrono>

//#include "lock.h"

using namespace std;

typedef struct trans_node {
	int value;
} TransNode;

typedef struct {
	int trans_no;
    int item_size;
    int item_code[1024];
} Transaction;

typedef struct {
	int item_no;
    int freq;
    int trans_array_size;
    int trans_array[128];
} Item;

typedef struct {
    int freq;
    int item_set_size;
	int item_set_code[16];
    int trans_array_size;
    int trans_array[16];

    /* the indices of previous sets */
    int set1_index;
    int set2_index;
    
    bool pruned;
} ItemSet;

typedef struct {
    int freq;
    int count;
    int *item_code_array;
} SupportCount;

#define TRANS_NUM 1000
#define ITEM_NUM 2000
#define NUM_THREADS 1
#define BLOCK_SIZE 1


__global__ 
void item_freq_count(int num_trans, Transaction *transArray, Item* itemArray)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = gridDim.x*blockDim.x;
    int i = tid;
    while ( i < num_trans) {
        int item_size = transArray[i].item_size;                 
        for (int j = 0; j < item_size; j++) {
            int item_code = transArray[i].item_code[j];
            //itemArray[item_code].freq++;
            atomicAdd(&(itemArray[item_code].freq), 1);
            /* push the transaction to the item struct */
            int _idx = atomicAdd(&(itemArray[item_code].trans_array_size), 1);
            itemArray[item_code].trans_array[_idx] = i;
        }
        i += num_threads;
    }
}

__global__
void select_with_min_support(int num_items, Item* itemArray, int min_support, ItemSet* itemsetArray, int* globalIdx)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = gridDim.x * blockDim.x;
    int i = tid;
    while (i < num_items) { 
        if (itemArray[i].freq >= min_support) {
            /* get a place in itemsetArray */
            int _idx = atomicAdd(globalIdx, 1);
            itemsetArray[_idx].freq = itemArray[i].freq;
            itemsetArray[_idx].item_set_size = 1;
            itemsetArray[_idx].item_set_code[0] = itemArray[i].item_no;
            itemsetArray[_idx].trans_array_size = itemArray[i].trans_array_size;
            memcpy(itemsetArray[_idx].trans_array, itemArray[i].trans_array, itemArray[i].trans_array_size*sizeof(int));
        }
        i += num_threads;
    }
}


__device__
bool alreadyHasTrans(ItemSet* _item_set, int trans_no)
{
    bool has = false;
    for (int i = 0; i < _item_set->trans_array_size; i++ ) {
        if (_item_set->trans_array[i] == trans_no) {
            has = true;
            break;
        }
    }
    return has;
}

/* search for transactions in the previous itemset, updating the transaction records
    and returning the count */
__device__
int find_support_count_for_itemset(ItemSet* candidate_itemset, ItemSet* checked_itemset, Transaction* trans_array)
{
    int count = 0;
    //printf("checked item set trans size %d\n", checked_itemset->trans_array_size);
    for (int i = 0; i < checked_itemset->trans_array_size; i++) {
        int trans_idx = checked_itemset->trans_array[i];
        Transaction* trans = &(trans_array[trans_idx]);
        bool itemset_found = true;
        int trans_no = -1;
        for (int j = 0; j < candidate_itemset->item_set_size; j++) {
            int target_item_code = candidate_itemset->item_set_code[j];
            bool single_item_found = false;
            for (int k = 0; k < trans->item_size; k++) {
                if (target_item_code == trans->item_code[k] && 
                    !alreadyHasTrans(candidate_itemset, trans_idx)) {
                    single_item_found = true; 
                    trans_no = trans_idx;
                    break;
                }
            }
            itemset_found &= single_item_found;
        }
        if (itemset_found) {
            candidate_itemset->trans_array[candidate_itemset->trans_array_size++] = trans_no;
            count++;
        }
    }
    return count;
}

__global__
void find_support_count(int candidateSetSize, ItemSet* candidateSet, int* globalIdx, ItemSet* currSet, Transaction* trans_array, int min_support)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = gridDim.x * blockDim.x;
    int i = tid;
    
    while (i < (candidateSetSize)) {
        int set1_idx = candidateSet[i].set1_index;
        int set2_idx = candidateSet[i].set2_index;
        int count1 = find_support_count_for_itemset(&(candidateSet[i]), &(currSet[set1_idx]), trans_array);
        int count2 = find_support_count_for_itemset(&(candidateSet[i]), &(currSet[set2_idx]), trans_array);
        
        candidateSet[i].freq = count1 + count2;
        /* check with minimum spport */ 
        if ((count1 + count2) >= min_support) {
            //int _global_idx = atomicAdd(globalIdx, 1);
            candidateSet[i].pruned = false;
        }
        else {
            candidateSet[i].pruned = true;
            //candidateSet[i].freq = -1;
        }
        i += num_threads;
    }
    
    /* block-level barrier */
    //__syncthreads();
}

int itemcodeComp(const void* a, const void* b)
{
    return (*(int*)a - *(int*)b);
}

int itemsetComp(const void* a, const void* b)
{
    ItemSet* set_a = (ItemSet*)(a);
    ItemSet* set_b = (ItemSet*)(b);
    int size = set_a->item_set_size;
    for (int i = 0; i < size; i++) {
        if (set_a->item_set_code[i] > set_b->item_set_code[i]) {
            return 1;
        }
        else if (set_a->item_set_code[i] < set_b->item_set_code[i]) {
            return -1;
        }
    }
    return 0;
}

bool hasTheItemSet(std::set<ItemSet*>& itemsets_set, ItemSet* checked_set) 
{
    for (auto it = itemsets_set.begin(); it != itemsets_set.end(); it++) {
        if ((*it)->item_set_size != checked_set->item_set_size) continue;
        if (memcmp((*it)->item_set_code, checked_set->item_set_code, checked_set->item_set_size*sizeof(int)) == 0) {
            return true; 
        }
    }
    return false;
}

int find_last_eq_class_item(int array_size, ItemSet* itemset_array, int base_pos, int start_pos, int cardinality)
{
    ItemSet* base_item_set = &(itemset_array[base_pos]);
    int last_pos = base_pos;
    
    if (cardinality < 2) {
        return -1;
    }

    for (int i = start_pos; i < array_size; i++) {
        ItemSet* check_item_set = &(itemset_array[i]);
        for (int j = 0; j < cardinality-1; j++) {
            if (base_item_set->item_set_code[j] != check_item_set->item_set_code[j]) {
                goto last_pos_ret; 
            }
        }
        last_pos = i;
    }

last_pos_ret:
    return last_pos;
}

void* genNextItemSetArray(int itemset_array_size, ItemSet* curr_itemset_array, int nextCardinality, int* nextSize)
{
    int _arr_size = itemset_array_size;
    int new_idx = 0;
    if (itemset_array_size <= 0) {
        return NULL;
    }
    
    assert(nextCardinality-1 == curr_itemset_array[0].item_set_size);
    
    ItemSet* next_set = NULL;
    
    if (nextCardinality == 2) {
        int next_size = (_arr_size*(_arr_size-1)) / 2;
        next_set = (ItemSet*)malloc(next_size*sizeof(ItemSet));
        assert(next_set != NULL);
        memset(next_set, 0, next_size*sizeof(ItemSet));
        for (int i = 0; i < _arr_size-1; i++) {
            for (int j = i+1; j < _arr_size; j++) {
                /* set up new itemset */
                next_set[new_idx].item_set_size = nextCardinality;
                next_set[new_idx].item_set_code[0] = curr_itemset_array[i].item_set_code[0];
                next_set[new_idx].item_set_code[1] = curr_itemset_array[j].item_set_code[0];
                
                /* store the indices */
                next_set[new_idx].set1_index = i;
                next_set[new_idx].set2_index = j;

                new_idx++;
            }
        }
        *nextSize = next_size;
    }
    else {
        int i = 0;
        vector< pair<int,int> > ranges_vec;
        while (i < itemset_array_size) {
            int j = find_last_eq_class_item(itemset_array_size, curr_itemset_array, i, i+1, nextCardinality-1);
            if ( (j != -1) && (i != j) ) {
                ranges_vec.push_back(make_pair(i,j));
            }
            i = j+1; 
        }
       
        auto pairSum = [](vector< pair<int,int> >& _vec) {
            int sum = 0;
            for (int i = 0; i < _vec.size(); i++) {
                int _size = (_vec[i].second-_vec[i].first+1);
                sum += (_size*(_size-1)/2);
            }
            return sum;
        };
        /* allocate next level item set memory */ 
        int next_size = pairSum(ranges_vec);
        if (next_size <= 0 || next_size > 1024*1024) {
            return NULL;
        }

        set<ItemSet*> itemsets_set;
        next_set = (ItemSet*)malloc((size_t)next_size*(size_t)sizeof(ItemSet));
        assert(next_set != NULL);
        memset(next_set, 0, next_size*sizeof(ItemSet));
        for (auto range : ranges_vec) {
            /* the priori nextCardinality-2 items should be the same */
            for (int start_pos = range.first; start_pos <= range.second-1; start_pos++) {
                for (int end_pos = start_pos+1; end_pos <= range.second; end_pos++) {
                    /* set up new itemset */
                    next_set[new_idx].item_set_size = nextCardinality;
                    
                    memcpy(next_set[new_idx].item_set_code,
                           curr_itemset_array[start_pos].item_set_code,
                           curr_itemset_array[start_pos].item_set_size*sizeof(int));
                    
                    next_set[new_idx].item_set_code[nextCardinality-1] = curr_itemset_array[end_pos].item_set_code[nextCardinality-2];
                    
                    /*
                    if (hasTheItemSet(itemsets_set, &(next_set[new_idx]))) {
                        next_size--;
                        continue;
                    }
                    itemsets_set.insert(&(next_set[new_idx]));
                    */

                    /* store the indices */
                    next_set[new_idx].set1_index = start_pos;
                    next_set[new_idx].set2_index = end_pos;
    
                    new_idx++; 
                }
            }
        }
        *nextSize = next_size;
    }

    return (void*)next_set;
}

int main(int argc, char *argv[]) 
{
    fstream fs;
    string line;
    unordered_map<string, int> item_code_map;
    unordered_map<int, int> transaction_map;
    vector<SupportCount> support_count_vec;

    int trans_count = 0;    /* number of transactions */
    int item_count = 0;     /* number of unique items */
    int min_support = 6;    /* mininum supoort of items */
    
    /* profiling */
    size_t memory_use = 0, max_memory_use = 0;
    size_t dev_memory_use = 0, max_dev_memory_use = 0;

    /* args */
    int opt;
    int num_threads = 0;
    int block_size = 0;
    int item_num = ITEM_NUM;
    const char *optstr = "n:b:i:";
    while ((opt = getopt(argc, argv, optstr)) != -1) {
        switch (opt) {
            case 'n':
                num_threads = atoi(optarg);
                break;
            case 'b':
                block_size = atoi(optarg);
                break;
            case 'i':
                item_num = atoi(optarg);
                break;
        }
    }
    printf("num threads %d, block size %d\n", num_threads, block_size); 

    Transaction *transArray = (Transaction*)malloc(TRANS_NUM*sizeof(Transaction));
    memset(transArray, 0, TRANS_NUM*sizeof(Transaction));
    memory_use += TRANS_NUM*sizeof(Transaction);

    /* read from the file */
    //fs.open("test.csv", ios::in);
    fs.open("data.csv", ios::in);
    while (getline(fs, line)) {
        if (line.size() == 0) continue;
        /* get transaction number */
        ssize_t pos = line.find(",");
        int trans_no = atoi(line.substr(0, pos).c_str());
        ssize_t pos2 = line.find(",", pos+1);
        string item = line.substr(pos+1, pos2-pos-1);
        
        /* find item number */
        if (item_code_map.find(item) == item_code_map.end()) {
            item_code_map[item] = item_count++;
            //printf("Item Count :%d -> %s\n", item_count, item.c_str());
        }
        /* find transaction number */
        if (transaction_map.find(trans_no) == transaction_map.end()) {
            transArray[trans_count].trans_no = trans_count;
            transArray[trans_count].item_code[transArray[trans_count].item_size++] = item_code_map[item];
            transaction_map[trans_no] = trans_count;
            trans_count++;
            //printf("Transaction Count :%d -> %d\n", trans_count, trans_no);
        }
        //else 
        {
            int _idx = transaction_map[trans_no]; 
            auto checkItemExist = [](Transaction* _tr, int _code) -> bool
            {
                bool ret = false;
                for (int idx = 0; idx < _tr->item_size; idx++) {
                    if (_tr->item_code[idx] == _code) return true;
                }
                return ret;
            };
            //if (!checkItemExist(&(transArray[_idx]), item_code_map[item])) 
            transArray[_idx].item_code[transArray[_idx].item_size++] = item_code_map[item];
        }

        if (trans_count >= TRANS_NUM) break;
        if (item_count >= item_num) break;
    }
    fs.close();
    
    printf("Item Count: %d, Transaction Count: %d\n", item_count, trans_count);
   
    size_t total = 0;
    auto begin = chrono::high_resolution_clock::now();

    /* sort item code array for each transaction */
    for (int _tr_idx = 0; _tr_idx < trans_count; _tr_idx++) {
        qsort(transArray[_tr_idx].item_code, transArray[_tr_idx].item_size, sizeof(int), itemcodeComp);
        int glb_i = 0;
        int i;
        for (i = 0; i < transArray[_tr_idx].item_size-1; i++) {
            while (i < transArray[_tr_idx].item_size && transArray[_tr_idx].item_code[i] == transArray[_tr_idx].item_code[i+1]) {
                i++;
            }
            transArray[_tr_idx].item_code[glb_i++] = transArray[_tr_idx].item_code[i];
        }
        if (i == transArray[_tr_idx].item_size-1) {
            transArray[_tr_idx].item_code[glb_i++] = transArray[_tr_idx].item_code[i-1];
        }
        transArray[_tr_idx].item_size = glb_i;
    }
    
    auto end = chrono::high_resolution_clock::now();
    total += chrono::duration_cast<chrono::milliseconds>(end-begin).count();
 
    auto printTrans = [](int _arr_size, Transaction* _trans_array)
    {
        for (int _tr_idx = 0; _tr_idx < _arr_size; _tr_idx++) {
            printf("Transaction %d:\n", _trans_array[_tr_idx].trans_no);
            for (int _it_idx = 0; _it_idx < _trans_array[_tr_idx].item_size; _it_idx++) {
                printf("\t Item %d\n", _trans_array[_tr_idx].item_code[_it_idx]); 
            }   
        }
    };
    //printTrans(trans_count, transArray);

    begin = chrono::high_resolution_clock::now();

    Item *itemArray = (Item*)malloc(item_count*sizeof(Item));
    memset(itemArray, 0, item_count*sizeof(Item));
    memory_use += item_count*sizeof(Item);
    for (int i = 0; i < item_count; i++) {
        itemArray[i].item_no = i;
    }
    
    /* request cuda memory */
    Transaction *dev_transArray = NULL;
    hipMalloc(&dev_transArray, TRANS_NUM*sizeof(Transaction));
    hipMemcpy(dev_transArray, transArray, TRANS_NUM*sizeof(Transaction), hipMemcpyHostToDevice);
    dev_memory_use += TRANS_NUM*sizeof(Item);
    
    Item *dev_itemArray = NULL;
    hipMalloc(&dev_itemArray, item_count*sizeof(Item));
    hipMemcpy(dev_itemArray, itemArray, item_count*sizeof(Item), hipMemcpyHostToDevice);
    dev_memory_use += item_count*sizeof(Item);
    
    /* calculate single item frequency */
    int num_threads_per_block = (num_threads < block_size) ? num_threads : block_size;
    //dim3 gridSize(num_threads/block_size);
    dim3 gridSize(NUM_THREADS/BLOCK_SIZE);
    dim3 blockSize(BLOCK_SIZE);
    item_freq_count<<<gridSize, blockSize>>>(trans_count, dev_transArray, dev_itemArray);

    /* copy the results back to host */
    hipMemcpy(itemArray, dev_itemArray, item_count*sizeof(Item), hipMemcpyDeviceToHost);
    
    /* sort transaction array for each item */
    for (int _it_idx = 0; _it_idx < item_count; _it_idx++) {
        qsort(itemArray[_it_idx].trans_array, itemArray[_it_idx].trans_array_size, sizeof(int), itemcodeComp);
        int glb_i = 0;
        int i;
        for (i = 0; i < itemArray[_it_idx].trans_array_size-1; i++) {
            while (i < itemArray[_it_idx].trans_array_size && itemArray[_it_idx].trans_array[i] == itemArray[_it_idx].trans_array[i+1]) {
                i++;
            }
            itemArray[_it_idx].trans_array[glb_i++] = itemArray[_it_idx].trans_array[i];
        }
        if (i == itemArray[_it_idx].trans_array_size-1) {
            itemArray[_it_idx].trans_array[glb_i++] = itemArray[_it_idx].trans_array[i-1];
        }
        itemArray[_it_idx].trans_array_size = glb_i;
    }
    
    end = chrono::high_resolution_clock::now();
    total += chrono::duration_cast<chrono::milliseconds>(end-begin).count();

    /* check point of transposed database */
    auto printItems = [](int _arr_size, Item* _item_array)
    {
        for (int _it_idx = 0; _it_idx < _arr_size; _it_idx++) {
            printf("Item %d (freq %d):\n", _item_array[_it_idx].item_no, _item_array[_it_idx].freq);
            for (int _tr_idx = 0; _tr_idx < _item_array[_it_idx].trans_array_size; _tr_idx++) {
                printf("\t Transaction %d\n", _item_array[_it_idx].trans_array[_tr_idx]); 
            }   
        }
    };
    //printItems(item_count, itemArray);
    
    begin = chrono::high_resolution_clock::now();
 
    /* start to prune */
    int globalIdx = 0;
    int *dev_globalIdx = NULL;
    hipMalloc(&dev_globalIdx, sizeof(int));
    hipMemcpy(dev_globalIdx, &globalIdx, sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(dev_itemArray, itemArray, item_count*sizeof(Item), hipMemcpyHostToDevice);

    ItemSet *itemsetArray = (ItemSet*)malloc(item_count*sizeof(ItemSet));
    memset(itemsetArray, 0, item_count*sizeof(ItemSet));
    memory_use += item_count*sizeof(ItemSet);
    
    ItemSet *dev_itemsetArray = NULL;
    hipMalloc(&dev_itemsetArray, item_count*sizeof(ItemSet));
    hipMemcpy(dev_itemsetArray, itemsetArray, item_count*sizeof(ItemSet), hipMemcpyHostToDevice);
    dev_memory_use += item_count*sizeof(ItemSet);
    
    /* kernel doing selection for single item with minimum support */
    select_with_min_support<<<gridSize, blockSize>>>(item_count, dev_itemArray, min_support, dev_itemsetArray, dev_globalIdx);

    hipMemcpy(itemsetArray, dev_itemsetArray, item_count*sizeof(ItemSet), hipMemcpyDeviceToHost);
    hipMemcpy(&globalIdx, dev_globalIdx, sizeof(int), hipMemcpyDeviceToHost);
   
    free(itemArray);
    
    end = chrono::high_resolution_clock::now();
    total += chrono::duration_cast<chrono::milliseconds>(end-begin).count();

    /* check point of transposed database */
    auto printItemSet = [](int _arr_size, ItemSet* _itemset_array)
    {
        for (int _it_idx = 0; _it_idx < _arr_size; _it_idx++) {
            printf("ItemSet %d (size %d):\n", _it_idx, _itemset_array[_it_idx].item_set_size);
            for (int i = 0; i < _itemset_array[_it_idx].item_set_size; i++) {
                printf("\tItem %d", _itemset_array[_it_idx].item_set_code[i]);
            }
            printf("\n");
            for (int i = 0; i < _itemset_array[_it_idx].trans_array_size; i++) {
                printf("\tTransaction %d", _itemset_array[_it_idx].trans_array[i]);
            }  
            printf("\n");
            printf("\tSet Index (%d,%d)\n", _itemset_array[_it_idx].set1_index, _itemset_array[_it_idx].set2_index);
        }
    };
    //printItemSet(globalIdx, itemsetArray);

 
    /* Record in Support Count */
    auto sc_record_func = [](vector<SupportCount>& vec, int itemset_count, ItemSet* itemset_array)
    {
        for (int is_idx = 0; is_idx < itemset_count; is_idx++) {
            SupportCount sc;
            sc.freq = itemset_array[is_idx].freq;
            sc.count = itemset_array[is_idx].item_set_size;
            sc.item_code_array = (int*)malloc(sc.count * sizeof(int));
            memcpy(sc.item_code_array, itemset_array[is_idx].item_set_code, sc.count*sizeof(int));
            qsort(sc.item_code_array, sc.count, sizeof(int), itemcodeComp);
            vec.push_back(sc);
        }
    };   
    
    sc_record_func(support_count_vec, item_count, itemsetArray);

    /* Now we get the transposed database that every item set with size 1 has a corresponding list of transactions */
    /* Generate itemset with size 2 */
    
    int cardinality = 2;
    int currSetSize = globalIdx;
    int candidateSetSize = 0;
    int *dev_candidateSetSize = NULL;
    ItemSet* currSet = itemsetArray;
    ItemSet* dev_currSet = NULL;
    ItemSet* candidateSet = NULL;
    ItemSet* dev_candidateSet = NULL;

    hipMalloc(&dev_candidateSetSize, sizeof(int));
    
    //cudaMalloc(&dev_currSet, currSetSize*sizeof(ItemSet));
    //cudaMemcpy(dev_currSet, currSet, currSetSize*sizeof(ItemSet), cudaMemcpyHostToDevice);

    while (true) {
        candidateSet = (ItemSet*)genNextItemSetArray(currSetSize, currSet, cardinality, &candidateSetSize);
        if (candidateSetSize == 0 || candidateSet == NULL) {
            break;
        }
        assert(candidateSet != NULL);          
        
        printf("\n\n Next candidate size is %d\n", candidateSetSize);
        memory_use += candidateSetSize*sizeof(ItemSet);
        
        begin = chrono::high_resolution_clock::now();

        /* allocate GPU kernel memory */
        hipMemcpy(dev_candidateSetSize, &candidateSetSize, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_globalIdx, &globalIdx, sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&dev_currSet, currSetSize*sizeof(ItemSet));
        assert(dev_currSet != NULL);
        hipMemcpy(dev_currSet, currSet, currSetSize*sizeof(ItemSet), hipMemcpyHostToDevice);
        hipMalloc(&dev_candidateSet, candidateSetSize*sizeof(ItemSet));
        hipMemcpy(dev_candidateSet, candidateSet, candidateSetSize*sizeof(ItemSet), hipMemcpyHostToDevice);        
        
        dev_memory_use += currSetSize*sizeof(ItemSet);
        dev_memory_use += candidateSetSize*sizeof(ItemSet);

        /* launch the kernel */
        dim3 gSize(NUM_THREADS/BLOCK_SIZE);
        dim3 bSize(BLOCK_SIZE);
        find_support_count<<<gSize, bSize>>>(candidateSetSize, 
                                                     dev_candidateSet, 
                                                     dev_globalIdx, 
                                                     dev_currSet, 
                                                     dev_transArray,
                                                     min_support);

        /* copy the result back */
        hipMemcpy(candidateSet, dev_candidateSet, candidateSetSize*sizeof(ItemSet), hipMemcpyDeviceToHost);
        
        end = chrono::high_resolution_clock::now();
        total += chrono::duration_cast<chrono::milliseconds>(end-begin).count(); 
        
        /* prune if freq == -1 */
        int _glb_set_idx = 0;
        for (int set_idx = 0; set_idx < candidateSetSize; set_idx++) {
            if (!candidateSet[set_idx].pruned) {
                //printf("---prune candidate %d freq %d\n", set_idx, candidateSet[set_idx].freq);
                memcpy(&(candidateSet[_glb_set_idx++]), &(candidateSet[set_idx]), sizeof(ItemSet));
            }
        }
        //candidateSetSize = _glb_set_idx;
        //printItemSet(_glb_set_idx, candidateSet); 

        /* Make statistics for support count */
        sc_record_func(support_count_vec, candidateSetSize, candidateSet);

        /* update the parameters and free previously used memory */
        free(currSet);
        hipFree(dev_currSet);
        
        cardinality++;
        currSet = candidateSet;
        currSetSize = candidateSetSize;
        //dev_currSet = dev_candidateSet; 
        hipFree(dev_candidateSet);
        globalIdx = 0;

        if (_glb_set_idx <= 1) {
            break;
        }
        
        max_memory_use = (max_memory_use < memory_use) ? memory_use : max_memory_use;
        max_dev_memory_use = (max_dev_memory_use < dev_memory_use) ? dev_memory_use : max_dev_memory_use;
        memory_use -= currSetSize*sizeof(ItemSet);
        dev_memory_use -= candidateSetSize*sizeof(ItemSet);
    }
    
    /* final result */
    //printItemSet(currSetSize, currSet); 
    
    /* Finally generate association rules */
    auto get_support_count = [](vector<SupportCount>& vec, ItemSet* itemset)->int
    {
        int _size = itemset->item_set_size;
        for (auto sc : vec) {
            if (sc.count != _size) continue;
            if (memcmp(itemset->item_set_code, sc.item_code_array, _size*sizeof(int)) != 0) continue;
            return sc.freq;
        }
        return 0;
    };
    
    function<void(ItemSet*, int, int, int, ItemSet*, vector<SupportCount>&)> get_rules_per_size;
    get_rules_per_size = [&get_support_count, &get_rules_per_size](ItemSet* sub_itemset, int array_index, int size, int start_pos, ItemSet* itemset, vector<SupportCount>& vec)
    {
        sub_itemset->item_set_code[array_index] = itemset->item_set_code[start_pos];
        if (array_index+1 == size) {
            int _support_count = get_support_count(vec, sub_itemset);
            /* now we can calculate the confidence */
            if (_support_count == 0) return;
            //printf("freq %f, s_count %f\n", (float)(itemset->freq), (float)(_support_count));
            float confidence =  (float)(itemset->freq) / (float)(_support_count);
            
            //printf("-----------------Association Rules--------------------\n");
            
            //printf("Items: \n");
            //for (int i = 0; i < size; i++) printf("\tItem %d\t", sub_itemset->item_set_code[i]);
            //printf("\nBase: \n");
            //for (int i = 0; i < itemset->item_set_size; i++) printf("\tItem %d\t", itemset->item_set_code[i]);
            //printf("\n\n=====  Confidence %f =====\n", confidence);

            //printf("------------------------------------------------------\n");
            
            return;
        }
        for (int next_pos = start_pos+1; next_pos < itemset->item_set_size; next_pos++) {
            get_rules_per_size(sub_itemset, array_index+1, size, next_pos, itemset, vec);
        }
    };

    
    auto getRules = [&get_rules_per_size](ItemSet* itemset, int size, vector<SupportCount>& vec)
    {
        //int *_code_array = (int*)malloc(size*sizeof(int));
        ItemSet *sub_itemset = (ItemSet*)malloc(sizeof(ItemSet));
        memset(sub_itemset, 0, sizeof(ItemSet));
        sub_itemset->item_set_size = size;
        
        int array_index = 0;
        for (int start_pos = 0; start_pos < itemset->item_set_size; start_pos++) {
            get_rules_per_size(sub_itemset, array_index, size, start_pos, itemset, vec);
        }

        free(sub_itemset);
    };

    for (int idx = 0; idx < currSetSize; idx++) {
        ItemSet* item_set = &currSet[idx];
        for (int _size = 1; _size <= item_set->item_set_size; _size++) {
            //getRules(item_set, _size, support_count_vec);
        }
    }
    
    hipProfilerStop();    
 
    printf("Sumary : Item Count %d --- Trans Count %d\nExec Time %llu ms\n", item_count, trans_count, total);
    printf("\t CPU memory max usage : %llu bytes\n", max_memory_use);
    printf("\t GPU memory max usage : %llu bytes\n", max_dev_memory_use);

    return 0;
}
